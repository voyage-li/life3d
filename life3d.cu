/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号: SA24011176
 * 姓名: 李远航
 * 邮箱: voyage@mail.ustc.edu.cn
 ------------------------------------------------*/

#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>

#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]

using std::cin, std::cout, std::endl;
using std::ifstream, std::ofstream;

// 存活细胞数
int population(int N, char* universe) {
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

// 打印世界状态
void print_universe(int N, char* universe) {
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++) {
        for (int y = 0; y < N; y++) {
            for (int z = 0; z < N; z++) {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}

// 核心计算代码，将世界向前推进T个时刻
__global__ void life3d_kernel(int N, char* universe, char* next) {
    __shared__ char shared_universe[10][10][10];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= N || y >= N || z >= N)
        return;

    int lx = threadIdx.x + 1;
    int ly = threadIdx.y + 1;
    int lz = threadIdx.z + 1;

    shared_universe[lx][ly][lz] = AT(x, y, z);

    if (lx == 1)
        shared_universe[0][ly][lz] = AT((x - 1 + N) % N, y, z);
    if (lx == blockDim.x)
        shared_universe[blockDim.x + 1][ly][lz] = AT((x + 1) % N, y, z);
    if (ly == 1)
        shared_universe[lx][0][lz] = AT(x, (y - 1 + N) % N, z);
    if (ly == blockDim.y)
        shared_universe[lx][blockDim.y + 1][lz] = AT(x, (y + 1) % N, z);
    if (lz == 1)
        shared_universe[lx][ly][0] = AT(x, y, (z - 1 + N) % N);
    if (lz == blockDim.z)
        shared_universe[lx][ly][blockDim.z + 1] = AT(x, y, (z + 1) % N);

    if (lx == 1 && ly == 1)
        shared_universe[0][0][lz] = AT((x - 1 + N) % N, (y - 1 + N) % N, z);
    if (lx == 1 && ly == blockDim.y)
        shared_universe[0][blockDim.y + 1][lz] = AT((x - 1 + N) % N, (y + 1) % N, z);
    if (lx == 1 && lz == 1)
        shared_universe[0][ly][0] = AT((x - 1 + N) % N, y, (z - 1 + N) % N);
    if (lx == 1 && lz == blockDim.z)
        shared_universe[0][ly][blockDim.z + 1] = AT((x - 1 + N) % N, y, (z + 1) % N);

    if (lx == blockDim.x && ly == 1)
        shared_universe[blockDim.x + 1][0][lz] = AT((x + 1) % N, (y - 1 + N) % N, z);
    if (lx == blockDim.x && ly == blockDim.y)
        shared_universe[blockDim.x + 1][blockDim.y + 1][lz] = AT((x + 1) % N, (y + 1) % N, z);
    if (lx == blockDim.x && lz == 1)
        shared_universe[blockDim.x + 1][ly][0] = AT((x + 1) % N, y, (z - 1 + N) % N);
    if (lx == blockDim.x && lz == blockDim.z)
        shared_universe[blockDim.x + 1][ly][blockDim.z + 1] = AT((x + 1) % N, y, (z + 1) % N);

    if (ly == 1 && lz == 1)
        shared_universe[lx][0][0] = AT(x, (y - 1 + N) % N, (z - 1 + N) % N);
    if (ly == 1 && lz == blockDim.z)
        shared_universe[lx][0][blockDim.z + 1] = AT(x, (y - 1 + N) % N, (z + 1) % N);
    if (ly == blockDim.y && lz == 1)
        shared_universe[lx][blockDim.y + 1][0] = AT(x, (y + 1) % N, (z - 1 + N) % N);
    if (ly == blockDim.y && lz == blockDim.z)
        shared_universe[lx][blockDim.y + 1][blockDim.z + 1] = AT(x, (y + 1) % N, (z + 1) % N);

    if (lx == 1 && ly == 1 && lz == 1)
        shared_universe[0][0][0] = AT((x - 1 + N) % N, (y - 1 + N) % N, (z - 1 + N) % N);
    if (lx == 1 && ly == 1 && lz == blockDim.z)
        shared_universe[0][0][blockDim.z + 1] = AT((x - 1 + N) % N, (y - 1 + N) % N, (z + 1) % N);
    if (lx == 1 && ly == blockDim.y && lz == 1)
        shared_universe[0][blockDim.y + 1][0] = AT((x - 1 + N) % N, (y + 1) % N, (z - 1 + N) % N);
    if (lx == 1 && ly == blockDim.y && lz == blockDim.z)
        shared_universe[0][blockDim.y + 1][blockDim.z + 1] = AT((x - 1 + N) % N, (y + 1) % N, (z + 1) % N);
    if (lx == blockDim.x && ly == 1 && lz == 1)
        shared_universe[blockDim.x + 1][0][0] = AT((x + 1) % N, (y - 1 + N) % N, (z - 1 + N) % N);
    if (lx == blockDim.x && ly == 1 && lz == blockDim.z)
        shared_universe[blockDim.x + 1][0][blockDim.z + 1] = AT((x + 1) % N, (y - 1 + N) % N, (z + 1) % N);
    if (lx == blockDim.x && ly == blockDim.y && lz == 1)
        shared_universe[blockDim.x + 1][blockDim.y + 1][0] = AT((x + 1) % N, (y + 1) % N, (z - 1 + N) % N);
    if (lx == blockDim.x && ly == blockDim.y && lz == blockDim.z)
        shared_universe[blockDim.x + 1][blockDim.y + 1][blockDim.z + 1] = AT((x + 1) % N, (y + 1) % N, (z + 1) % N);

    __syncthreads();

    int alive = 0;
    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            for (int dz = -1; dz <= 1; dz++) {
                if (dx == 0 && dy == 0 && dz == 0)
                    continue;
                alive += shared_universe[lx + dx][ly + dy][lz + dz];
            }
        }
    }
    int idx = x * N * N + y * N + z;
    if (shared_universe[lx][ly][lz] && (alive < 5 || alive > 7))
        next[idx] = 0;
    else if (!shared_universe[lx][ly][lz] && alive == 6)
        next[idx] = 1;
    else
        next[idx] = shared_universe[lx][ly][lz];
}

void life3d_run(int N, char* universe, int T) {
    char *d_universe, *d_next;
    hipMalloc(&d_universe, N * N * N);
    hipMalloc(&d_next, N * N * N);
    hipMemcpy(d_universe, universe, N * N * N, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (N + threadsPerBlock.z - 1) / threadsPerBlock.z);
    for (int t = 0; t < T; t++) {
        life3d_kernel<<<numBlocks, threadsPerBlock>>>(N, d_universe, d_next);
        hipDeviceSynchronize();

        char* temp = d_universe;
        d_universe = d_next;
        d_next = temp;
    }

    hipMemcpy(universe, d_universe, N * N * N, hipMemcpyDeviceToHost);

    hipFree(d_universe);
    hipFree(d_next);
}

// 读取输入文件
void read_file(char* input_file, char* buffer) {
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size)) {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char* output_file, char* buffer, int N) {
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file) {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char** argv) {
    // cmd args
    if (argc < 5) {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char* input_file = argv[3];
    char* output_file = argv[4];

    char* universe = (char*)malloc(N * N * N);
    read_file(input_file, universe);

    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    life3d_run(N, universe, T);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);

    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    free(universe);
    return 0;
}
