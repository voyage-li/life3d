/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号: SA24011176
 * 姓名: 李远航
 * 邮箱: voyage@mail.ustc.edu.cn
 ------------------------------------------------*/

#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>

#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]
#define AT_BOUNDARY(x, y, z) AT((x + N) % N, (y + N) % N, (z + N) % N)

using std::cin, std::cout, std::endl;
using std::ifstream, std::ofstream;

// 存活细胞数
int population(int N, char* universe) {
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

// 打印世界状态
void print_universe(int N, char* universe) {
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++) {
        for (int y = 0; y < N; y++) {
            for (int z = 0; z < N; z++) {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}

// 核心计算代码，将世界向前推进T个时刻
__global__ void life3d_kernel(int N, char* universe, char* next) {
    __shared__ char shared_universe[10][10][10];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= N || y >= N || z >= N)
        return;

    int lx = threadIdx.x + 1;
    int ly = threadIdx.y + 1;
    int lz = threadIdx.z + 1;

    shared_universe[lx][ly][lz] = AT(x, y, z);

    const int dx[3] = {-1, 0, 1};
    const int dy[3] = {-1, 0, 1};
    const int dz[3] = {-1, 0, 1};
    const int sx[3] = {0, lx, (int)blockDim.x + 1};
    const int sy[3] = {0, ly, (int)blockDim.y + 1};
    const int sz[3] = {0, lz, (int)blockDim.z + 1};

    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            for (int k = 0; k < 3; k++) {
                if (i == 1 && j == 1 && k == 1)
                    continue;
                if ((i == 0 && lx != 1) || (i == 2 && lx != blockDim.x) || (j == 0 && ly != 1) || (j == 2 && ly != blockDim.y) || (k == 0 && lz != 1) || (k == 2 && lz != blockDim.z))
                    continue;
                if ((i == 0 && lx == 1) || (i == 2 && lx == blockDim.x) ||
                    (j == 0 && ly == 1) || (j == 2 && ly == blockDim.y) ||
                    (k == 0 && lz == 1) || (k == 2 && lz == blockDim.z)) {
                    shared_universe[sx[i]][sy[j]][sz[k]] = AT_BOUNDARY(x + dx[i], y + dy[j], z + dz[k]);
                }
            }
        }
    }

    __syncthreads();

    int alive = 0;
    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            for (int dz = -1; dz <= 1; dz++) {
                if (dx == 0 && dy == 0 && dz == 0)
                    continue;
                alive += shared_universe[lx + dx][ly + dy][lz + dz];
            }
        }
    }
    int idx = x * N * N + y * N + z;
    if (shared_universe[lx][ly][lz] && (alive < 5 || alive > 7))
        next[idx] = 0;
    else if (!shared_universe[lx][ly][lz] && alive == 6)
        next[idx] = 1;
    else
        next[idx] = shared_universe[lx][ly][lz];
}

void life3d_run(int N, char* universe, int T) {
    char *d_universe, *d_next;
    hipMalloc(&d_universe, N * N * N);
    hipMalloc(&d_next, N * N * N);
    hipMemcpy(d_universe, universe, N * N * N, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (N + threadsPerBlock.z - 1) / threadsPerBlock.z);
    for (int t = 0; t < T; t++) {
        life3d_kernel<<<numBlocks, threadsPerBlock>>>(N, d_universe, d_next);
        hipDeviceSynchronize();

        char* temp = d_universe;
        d_universe = d_next;
        d_next = temp;
    }

    hipMemcpy(universe, d_universe, N * N * N, hipMemcpyDeviceToHost);

    hipFree(d_universe);
    hipFree(d_next);
}

// 读取输入文件
void read_file(char* input_file, char* buffer) {
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size)) {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char* output_file, char* buffer, int N) {
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file) {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char** argv) {
    // cmd args
    if (argc < 5) {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char* input_file = argv[3];
    char* output_file = argv[4];

    char* universe = (char*)malloc(N * N * N);
    read_file(input_file, universe);

    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    life3d_run(N, universe, T);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);

    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    free(universe);
    return 0;
}
